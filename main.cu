#include "hip/hip_runtime.h"
#include <type_traits>
#include <iostream>
#include <numeric>
#include <vector>

#include <hip/hip_runtime.h>

#include "cuda/numeric"

template <cuda::thread_scope scope, typename data_type>
__global__ void perform_reduce (const data_type *array, unsigned int n, data_type *result)
{
  int value = cuda::reduce<scope>(array, array + n, data_type {}, [] __device__ (const data_type &a, const data_type &b) { return a + b; });

  if (threadIdx.x < n)
    result[threadIdx.x] = value;
}

template <typename data_type>
void expect_eq (
    const std::vector<data_type> input,
    const std::vector<data_type> &excected_output,
    unsigned int threads_per_block)
{
  data_type *device_input {};
  data_type *device_result {};

  hipMalloc (&device_input, input.size () * sizeof (data_type));
  hipMalloc (&device_result, input.size () * sizeof (data_type));

  hipMemcpy (device_input, input.data (), input.size () * sizeof (data_type), hipMemcpyHostToDevice);

  const int blocks_count = (input.size () + threads_per_block - 1) / threads_per_block;
  perform_reduce<cuda::thread_scope_warp><<<blocks_count, threads_per_block>>>(device_input, input.size (), device_result);

  std::vector<data_type> output (input.size (), data_type {});
  hipMemcpy (output.data (), device_result, input.size () * sizeof (data_type), hipMemcpyDeviceToHost);

  for (size_t i = 0; i < input.size (); i++)
    if (excected_output[i] != output[i])
      throw std::runtime_error ("Error: unexpected value at " + std::to_string (i));

  hipFree (device_input);
  hipFree (device_result);
}

template <typename data_type>
void perform_single_value_warp_size_test (const data_type &magical_value)
{
  const int warpSize = 32;
  std::vector<data_type> iv32 (warpSize, data_type {});
  std::vector<data_type> ov32 (warpSize, magical_value);

  for (int lane = 0; lane < warpSize; lane++)
  {
    iv32[lane] = magical_value;
    expect_eq(iv32, ov32, warpSize);
    iv32[lane] = data_type {};
  }
}

class user_type
{
  unsigned long long int x {};
  unsigned long long int y {};
public:
  user_type () = default;
  user_type (unsigned long long int x_arg, unsigned long long int y_arg) : x (x_arg), y (y_arg) {}

  friend __device__ user_type operator+ (const user_type &lhs, const user_type &rhs)
  {
    return user_type (lhs.x + rhs.x, lhs.y + rhs.y);
  }
};

static_assert(std::is_trivially_copyable<user_type>::value);

int main ()
{
  perform_single_value_warp_size_test(42);
  // perform_single_value_warp_size_test(user_type {4, 2});

  return 0;
}
